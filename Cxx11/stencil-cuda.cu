#include "hip/hip_runtime.h"

///
/// Copyright (c) 2013, Intel Corporation
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions
/// are met:
///
/// * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
/// * Redistributions in binary form must reproduce the above
///       copyright notice, this list of conditions and the following
///       disclaimer in the documentation and/or other materials provided
///       with the distribution.
/// * Neither the name of Intel Corporation nor the names of its
///       contributors may be used to endorse or promote products
///       derived from this software without specific prior written
///       permission.
///
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
/// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
/// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
/// COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
/// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
/// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
/// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
/// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
/// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
/// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.

//////////////////////////////////////////////////////////////////////
///
/// NAME:    Stencil
///
/// PURPOSE: This program tests the efficiency with which a space-invariant,
///          linear, symmetric filter (stencil) can be applied to a square
///          grid or image.
///
/// USAGE:   The program takes as input the linear
///          dimension of the grid, and the number of iterations on the grid
///
///                <progname> <iterations> <grid size>
///
///          The output consists of diagnostics to make sure the
///          algorithm worked, and of timing statistics.
///
/// FUNCTIONS CALLED:
///
///          Other than standard C functions, the following functions are used in
///          this program:
///          wtime()
///
/// HISTORY: - Written by Rob Van der Wijngaart, February 2009.
///          - RvdW: Removed unrolling pragmas for clarity;
///            added constant to array "in" at end of each iteration to force
///            refreshing of neighbor data in parallel versions; August 2013
///            C++11-ification by Jeff Hammond, May 2017.
///
//////////////////////////////////////////////////////////////////////

#include "prk_util.h"
#include "prk_cuda.h"
#include "stencil_cuda.hpp"

__global__ void nothing(const int n, const prk_float * in, prk_float * out)
{
    //printf("You are trying to use a stencil that does not exist.\n");
    //printf("Please generate the new stencil using the code generator.\n");
    // n will never be zero - this is to silence compiler warnings.
    //if (n==0) printf("in=%p out=%p\n", in, out);
    //abort();
}

__global__ void add(const int n, prk_float * in)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i<n) && (j<n)) {
        in[i*n+j] += (prk_float)1;
    }
}

int main(int argc, char* argv[])
{
  std::cout << "Parallel Research Kernels version " << PRKVERSION << std::endl;
  std::cout << "C++11/CUDA Stencil execution on 2D grid" << std::endl;

  prk::CUDA::info info;
  info.print();

  //////////////////////////////////////////////////////////////////////
  // Process and test input parameters
  //////////////////////////////////////////////////////////////////////

  int iterations, n, radius, tile_size;
  bool star = true;
  try {
      if (argc < 3) {
        throw "Usage: <# iterations> <array dimension> [<tile_size> <star/grid> <radius>]";
      }

      // number of times to run the algorithm
      iterations  = std::atoi(argv[1]);
      if (iterations < 1) {
        throw "ERROR: iterations must be >= 1";
      }

      // linear grid dimension
      n  = std::atoi(argv[2]);
      if (n < 1) {
        throw "ERROR: grid dimension must be positive";
      } else if (n > prk::get_max_matrix_size()) {
        throw "ERROR: grid dimension too large - overflow risk";
      }

      // default tile size for tiling of local transpose
      tile_size = 32;
      if (argc > 3) {
          tile_size = std::atoi(argv[3]);
          if (tile_size <= 0) tile_size = n;
          if (tile_size > n) tile_size = n;
          if (tile_size > 32) {
              std::cout << "Warning: tile_size > 32 may lead to incorrect results (observed for CUDA 9.0 on GV100).\n";
          }
      }

      // stencil pattern
      if (argc > 4) {
          auto stencil = std::string(argv[4]);
          auto grid = std::string("grid");
          star = (stencil == grid) ? false : true;
      }

      // stencil radius
      radius = 2;
      if (argc > 5) {
          radius = std::atoi(argv[5]);
      }

      if ( (radius < 1) || (2*radius+1 > n) ) {
        throw "ERROR: Stencil radius negative or too large";
      }
  }
  catch (const char * e) {
    std::cout << e << std::endl;
    return 1;
  }

  std::cout << "Number of iterations = " << iterations << std::endl;
  std::cout << "Grid size            = " << n << std::endl;
  std::cout << "Tile size            = " << tile_size << std::endl;
  std::cout << "Type of stencil      = " << (star ? "star" : "grid") << std::endl;
  std::cout << "Radius of stencil    = " << radius << std::endl;

  auto stencil = nothing;
  if (star) {
      switch (radius) {
          case 1: stencil = star1; break;
          case 2: stencil = star2; break;
          case 3: stencil = star3; break;
          case 4: stencil = star4; break;
          case 5: stencil = star5; break;
      }
  } else {
      switch (radius) {
          case 1: stencil = grid1; break;
          case 2: stencil = grid2; break;
          case 3: stencil = grid3; break;
          case 4: stencil = grid4; break;
          case 5: stencil = grid5; break;
      }
  }

  dim3 dimGrid(prk::divceil(n,tile_size),prk::divceil(n,tile_size),1);
  dim3 dimBlock(tile_size, tile_size, 1);
  info.checkDims(dimBlock, dimGrid);

  //////////////////////////////////////////////////////////////////////
  // Allocate space and perform the computation
  //////////////////////////////////////////////////////////////////////

  double stencil_time{0};

  const size_t nelems = (size_t)n * (size_t)n;
  const size_t bytes = nelems * sizeof(prk_float);
  prk_float * h_in;
  prk_float * h_out;
#ifndef __CORIANDERCC__
  prk::CUDA::check( hipHostMalloc((void**)&h_in, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_out, bytes) );
#else
  h_in = new prk_float[nelems];
  h_out = new prk_float[nelems];
#endif

  for (int i=0; i<n; i++) {
    for (int j=0; j<n; j++) {
      h_in[i*n+j]  = static_cast<prk_float>(i+j);
      h_out[i*n+j] = static_cast<prk_float>(0);
    }
  }

  // copy input from host to device
  prk_float * d_in;
  prk_float * d_out;
  prk::CUDA::check( hipMalloc((void**)&d_in, bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_out, bytes) );
  prk::CUDA::check( hipMemcpy(d_in, &(h_in[0]), bytes, hipMemcpyHostToDevice) );
  prk::CUDA::check( hipMemcpy(d_out, &(h_out[0]), bytes, hipMemcpyHostToDevice) );

  for (int iter = 0; iter<=iterations; iter++) {

    if (iter==1) stencil_time = prk::wtime();

    // Apply the stencil operator
    stencil<<<dimGrid, dimBlock>>>(n, d_in, d_out);

    // Add constant to solution to force refresh of neighbor data, if any
    add<<<dimGrid, dimBlock>>>(n, d_in);

#ifndef __CORIANDERCC__
    // silence "ignoring hipDeviceSynchronize for now" warning
    prk::CUDA::check( hipDeviceSynchronize() );
#endif
  }
  stencil_time = prk::wtime() - stencil_time;

  // copy output back to host
  prk::CUDA::check( hipMemcpy(&(h_out[0]), d_out, bytes, hipMemcpyDeviceToHost) );

#ifdef VERBOSE
  // copy input back to host - debug only
  prk::CUDA::check( hipMemcpy(&(h_in[0]), d_in, bytes, hipMemcpyDeviceToHost) );
#endif

  prk::CUDA::check( hipFree(d_out) );
  prk::CUDA::check( hipFree(d_in) );

  //////////////////////////////////////////////////////////////////////
  // Analyze and output results.
  //////////////////////////////////////////////////////////////////////

  // interior of grid with respect to stencil
  size_t active_points = static_cast<size_t>(n-2*radius)*static_cast<size_t>(n-2*radius);
  double norm = 0.0;
  for (int i=radius; i<n-radius; i++) {
    for (int j=radius; j<n-radius; j++) {
      norm += prk::abs(h_out[i*n+j]);
    }
  }
  norm /= active_points;

  // verify correctness
  const double epsilon = 1.0e-8;
  double reference_norm = 2.*(iterations+1.);
  if (prk::abs(norm-reference_norm) > epsilon) {
    std::cout << "ERROR: L1 norm = " << norm
              << " Reference L1 norm = " << reference_norm << std::endl;
    return 1;
  } else {
    std::cout << "Solution validates" << std::endl;
#ifdef VERBOSE
    std::cout << "L1 norm = " << norm
              << " Reference L1 norm = " << reference_norm << std::endl;
#endif
    const int stencil_size = star ? 4*radius+1 : (2*radius+1)*(2*radius+1);
    size_t flops = (2L*(size_t)stencil_size+1L) * active_points;
    auto avgtime = stencil_time/iterations;
    std::cout << "Rate (MFlops/s): " << 1.0e-6 * static_cast<double>(flops)/avgtime
              << " Avg time (s): " << avgtime << std::endl;
  }

  return 0;
}
