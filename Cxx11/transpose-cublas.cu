#include "hip/hip_runtime.h"
///
/// Copyright (c) 2013, Intel Corporation
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions
/// are met:
///
/// * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
/// * Redistributions in binary form must reproduce the above
///       copyright notice, this list of conditions and the following
///       disclaimer in the documentation and/or other materials provided
///       with the distribution.
/// * Neither the name of Intel Corporation nor the names of its
///       contributors may be used to endorse or promote products
///       derived from this software without specific prior written
///       permission.
///
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
/// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
/// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
/// COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
/// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
/// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
/// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
/// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
/// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
/// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.

//////////////////////////////////////////////////////////////////////
///
/// NAME:    transpose
///
/// PURPOSE: This program measures the time for the transpose of a
///          column-major stored matrix into a row-major stored matrix.
///
/// USAGE:   Program input is the matrix order and the number of times to
///          repeat the operation:
///
///          transpose <matrix_size> <# iterations> [tile size]
///
///          An optional parameter specifies the tile size used to divide the
///          individual matrix blocks for improved cache and TLB performance.
///
///          The output consists of diagnostics to make sure the
///          transpose worked and timing statistics.
///
/// HISTORY: Written by  Rob Van der Wijngaart, February 2009.
///          Converted to C++11 by Jeff Hammond, February 2016 and May 2017.
///
//////////////////////////////////////////////////////////////////////

#include "prk_util.h"
#include "prk_cuda.h"

#define CUBLAS_AXPY_BUG 1

int main(int argc, char * argv[])
{
  std::cout << "Parallel Research Kernels version " << PRKVERSION << std::endl;
  std::cout << "C++11/CUBLAS Matrix transpose: B = A^T" << std::endl;

  prk::CUDA::info info;
  info.print();

  //////////////////////////////////////////////////////////////////////
  // Read and test input parameters
  //////////////////////////////////////////////////////////////////////

  int iterations;
  int order;
  try {
      if (argc < 3) {
        throw "Usage: <# iterations> <matrix order>";
      }

      iterations  = std::atoi(argv[1]);
      if (iterations < 1) {
        throw "ERROR: iterations must be >= 1";
      }

      order = std::atoi(argv[2]);
      if (order <= 0) {
        throw "ERROR: Matrix Order must be greater than 0";
      } else if (order > prk::get_max_matrix_size()) {
        throw "ERROR: matrix dimension too large - overflow risk";
      }
  }
  catch (const char * e) {
    std::cout << e << std::endl;
    return 1;
  }

  std::cout << "Number of iterations = " << iterations << std::endl;
  std::cout << "Matrix order         = " << order << std::endl;

  hipblasHandle_t h;
  //prk::CUDA::check( cublasInit() );
  prk::CUDA::check( hipblasCreate(&h) );

  //////////////////////////////////////////////////////////////////////
  // Allocate space for the input and transpose matrix
  //////////////////////////////////////////////////////////////////////

  const size_t nelems = (size_t)order * (size_t)order;
  const size_t bytes = nelems * sizeof(double);

  double * h_a;
  double * h_b;
  prk::CUDA::check( hipHostMalloc((void**)&h_a, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_b, bytes) );

  // fill A with the sequence 0 to order^2-1 as doubles
  for (int j=0; j<order; j++) {
    for (int i=0; i<order; i++) {
      h_a[j*order+i] = order*j+i;
      h_b[j*order+i] = 0;
    }
  }

  // copy input from host to device
  double * d_a;
  double * d_b;
  prk::CUDA::check( hipMalloc((void**)&d_a, bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_b, bytes) );
  prk::CUDA::check( hipMemcpy(d_a, &(h_a[0]), bytes, hipMemcpyHostToDevice) );
  prk::CUDA::check( hipMemcpy(d_b, &(h_b[0]), bytes, hipMemcpyHostToDevice) );

#if CUBLAS_AXPY_BUG
  // We need a vector of ones because CUBLAS daxpy does not
  // correctly implement incx=0.
  double * h_o;
  prk::CUDA::check( hipHostMalloc((void**)&h_o, bytes) );
  for (int j=0; j<order; j++) {
    for (int i=0; i<order; i++) {
      h_o[j*order+i] = 1;
    }
  }
  double * d_o;
  prk::CUDA::check( hipMalloc((void**)&d_o, bytes) );
  prk::CUDA::check( hipMemcpy(d_o, &(h_o[0]), bytes, hipMemcpyHostToDevice) );
#endif

#ifdef USE_HOST_BUFFERS
  double p_a = h_a;
  double p_b = h_b;
#if CUBLAS_AXPY_BUG
  double p_o = h_o;
#endif
#else
  double * p_a = d_a;
  double * p_b = d_b;
#if CUBLAS_AXPY_BUG
  double * p_o = d_o;
#endif
#endif

  double trans_time{0};

  for (int iter = 0; iter<=iterations; iter++) {

    if (iter==1) trans_time = prk::wtime();

    double one(1);
    // B += trans(A) i.e. B = trans(A) + B
    prk::CUDA::check( hipblasDgeam(h,
                                  HIPBLAS_OP_T, HIPBLAS_OP_N,   // opA, opB
                                  order, order,               // m, n
                                  &one, p_a, order,           // alpha, A, lda
                                  &one, p_b, order,           // beta, B, ldb
                                  p_b, order) );              // C, ldc (in-place for B)

    // A += 1.0 i.e. A = 1.0 * 1.0 + A
#if CUBLAS_AXPY_BUG
    // THIS IS CORRECT
    prk::CUDA::check( hipblasDaxpy(h,
                      order*order,                // n
                      &one,                       // alpha
                      p_o, 1,                     // x, incx
                      p_a, 1) );                  // y, incy
#else
    // THIS IS BUGGY
    prk::CUDA::check( hipblasDaxpy(h,
                      order*order,                // n
                      &one,                       // alpha
                      &one, 0,                    // x, incx
                      p_a, 1) );                  // y, incy
#endif
    // (Host buffer version)
    // The performance is ~10% better if this is done every iteration,
    // instead of only once before the timer is stopped.
    prk::CUDA::check( hipDeviceSynchronize() );
  }
  trans_time = prk::wtime() - trans_time;

  // copy output back to host
  prk::CUDA::check( hipMemcpy(&(h_b[0]), d_b, bytes, hipMemcpyDeviceToHost) );

#if CUBLAS_AXPY_BUG
  prk::CUDA::check( hipFree(d_o) );
  prk::CUDA::check( hipHostFree(h_o) );
#endif

  prk::CUDA::check( hipFree(d_b) );
  prk::CUDA::check( hipFree(d_a) );

  prk::CUDA::check( hipHostFree(h_a) );

  prk::CUDA::check( hipblasDestroy(h) );
  //prk::CUDA::check( cublasShutdown() );

  //////////////////////////////////////////////////////////////////////
  /// Analyze and output results
  //////////////////////////////////////////////////////////////////////

  // TODO: replace with std::generate, std::accumulate, or similar
  const double addit = (iterations+1.) * (iterations/2.);
  double abserr(0);
  for (int j=0; j<order; j++) {
    for (int i=0; i<order; i++) {
      const size_t ij = (size_t)i*(size_t)order+(size_t)j;
      const size_t ji = (size_t)j*(size_t)order+(size_t)i;
      const double reference = static_cast<double>(ij)*(1.+iterations)+addit;
      abserr += prk::abs(h_b[ji] - reference);
    }
  }

#ifdef VERBOSE
  std::cout << "Sum of absolute differences: " << abserr << std::endl;
#endif

  const double epsilon = 1.0e-8;
  if (abserr < epsilon) {
    std::cout << "Solution validates" << std::endl;
    auto avgtime = trans_time/iterations;
    auto bytes = (size_t)order * (size_t)order * sizeof(double);
    std::cout << "Rate (MB/s): " << 1.0e-6 * (2L*bytes)/avgtime
              << " Avg time (s): " << avgtime << std::endl;
  } else {
#ifdef VERBOSE
    for (int i=0; i<order; i++) {
      for (int j=0; j<order; j++) {
        std::cout << "(" << i << "," << j << ") = " << h_a[i*order+j] << ", " << h_b[i*order+j] << "\n";
      }
    }
#endif
    std::cout << "ERROR: Aggregate squared error " << abserr
              << " exceeds threshold " << epsilon << std::endl;
    return 1;
  }

  prk::CUDA::check( hipHostFree(h_b) );

  return 0;
}


