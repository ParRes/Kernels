#include "hip/hip_runtime.h"
///
/// Copyright (c) 2018, Intel Corporation
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions
/// are met:
///
/// * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
/// * Redistributions in binary form must reproduce the above
///       copyright notice, this list of conditions and the following
///       disclaimer in the documentation and/or other materials provided
///       with the distribution.
/// * Neither the name of Intel Corporation nor the names of its
///       contributors may be used to endorse or promote products
///       derived from this software without specific prior written
///       permission.
///
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
/// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
/// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
/// COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
/// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
/// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
/// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
/// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
/// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
/// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.

//////////////////////////////////////////////////////////////////////
///
/// NAME:    dgemm
///
/// PURPOSE: This program tests the efficiency with which a dense matrix
///          dense multiplication is carried out
///
/// USAGE:   The program takes as input the matrix order,
///          the number of times the matrix-matrix multiplication
///          is carried out, and, optionally, a tile size for matrix
///          blocking
///
///          <progname> <# iterations> <matrix order> [<tile size>]
///
///          The output consists of diagnostics to make sure the
///          algorithm worked, and of timing statistics.
///
/// FUNCTIONS CALLED:
///
///          Other than OpenMP or standard C functions, the following
///          functions are used in this program:
///
///          wtime()
///
/// HISTORY: Written by Rob Van der Wijngaart, February 2009.
///          Converted to C++11 by Jeff Hammond, December, 2017.
///
//////////////////////////////////////////////////////////////////////

#include "prk_util.h"
#include "prk_cuda.h"

__global__ void init(unsigned order, double * A, double * B, double * C)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i<order) && (j<order)) {
      A[i*order+j] = i;
      B[i*order+j] = i;
      C[i*order+j] = 0;
    }
}

__global__ void init(unsigned order, double * C)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i<order) && (j<order)) {
      C[i*order+j] = 0;
    }
}

int main(int argc, char * argv[])
{
  std::cout << "Parallel Research Kernels version " << PRKVERSION << std::endl;
  std::cout << "C++11/CUBLAS Dense matrix-matrix multiplication: C += A x B" << std::endl;

  prk::CUDA::info info;
  info.print();

  //////////////////////////////////////////////////////////////////////
  /// Read and test input parameters
  //////////////////////////////////////////////////////////////////////

  int iterations;
  int order;
  int input_copy = 0;
  try {
      if (argc < 3) {
        throw "Usage: <# iterations> <matrix order> <copy input every iteration [0/1]>";
      }

      iterations  = std::atoi(argv[1]);
      if (iterations < 1) {
        throw "ERROR: iterations must be >= 1";
      }

      order = std::atoi(argv[2]);
      if (order <= 0) {
        throw "ERROR: Matrix Order must be greater than 0";
      } else if (order > std::floor(std::sqrt(INT_MAX))) {
        throw "ERROR: matrix dimension too large - overflow risk";
      }

      if (argc > 3) {
        input_copy = std::atoi(argv[3]);
      }
  }
  catch (const char * e) {
    std::cout << e << std::endl;
    return 1;
  }

  std::cout << "Number of iterations = " << iterations << std::endl;
  std::cout << "Matrix order         = " << order << std::endl;
  std::cout << "Input copy           = " << (input_copy ? "yes" : "no") << std::endl;

  hipblasHandle_t h;
  prk::CUDA::check( hipblasCreate(&h) );

  int tile_size = 32;
  dim3 dimGrid(prk::divceil(order,tile_size),prk::divceil(order,tile_size),1);
  dim3 dimBlock(tile_size, tile_size, 1);

  info.checkDims(dimBlock, dimGrid);

  //////////////////////////////////////////////////////////////////////
  // Allocate space for matrices
  //////////////////////////////////////////////////////////////////////

  double dgemm_time(0);

  const size_t nelems = (size_t)order * (size_t)order;
  const size_t bytes = nelems * sizeof(double);

  // host buffers
  double * h_a;
  double * h_b;
  double * h_c;
  prk::CUDA::check( hipHostMalloc((void**)&h_a, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_b, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_c, bytes) );

  // device buffers
  double * d_a;
  double * d_b;
  double * d_c;
  prk::CUDA::check( hipMalloc((void**)&d_a, bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_b, bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_c, bytes) );

  if (input_copy) {

    for (int i=0; i<order; ++i) {
      for (int j=0; j<order; ++j) {
         h_a[i*order+j] = i;
         h_b[i*order+j] = i;
      }
    }

    prk::CUDA::check( hipMemcpy(d_a, &(h_a[0]), bytes, hipMemcpyHostToDevice) );
    prk::CUDA::check( hipMemcpy(d_b, &(h_b[0]), bytes, hipMemcpyHostToDevice) );

    init<<<dimGrid, dimBlock>>>(order, d_c);

  } else {

    init<<<dimGrid, dimBlock>>>(order, d_a, d_b, d_c);

  }

  {
    for (auto iter = 0; iter<=iterations; iter++) {

      if (iter==1) dgemm_time = prk::wtime();

      if (input_copy) {
        prk::CUDA::check( hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice) );
        prk::CUDA::check( hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice) );
      }

      double one(1);
      prk::CUDA::check( hipblasDgemm(h,
                                    HIPBLAS_OP_N, HIPBLAS_OP_N, // opA, opB
                                    order, order, order,      // m, n, k
                                    &one,                     // alpha
                                    d_a, order,               // A, lda
                                    d_b, order,               // B, ldb
                                    &one,                     // beta
                                    d_c, order) );            // C, ldc
      prk::CUDA::check( hipDeviceSynchronize() );
    }
    dgemm_time = prk::wtime() - dgemm_time;
  }

  // copy output back to host
  prk::CUDA::check( hipMemcpy(&(h_c[0]), d_c, bytes, hipMemcpyDeviceToHost) );

  prk::CUDA::check( hipFree(d_c) );
  prk::CUDA::check( hipFree(d_b) );
  prk::CUDA::check( hipFree(d_a) );

  prk::CUDA::check( hipHostFree(h_a) );
  prk::CUDA::check( hipHostFree(h_b) );

  prk::CUDA::check( hipblasDestroy(h) );

  //////////////////////////////////////////////////////////////////////
  /// Analyze and output results
  //////////////////////////////////////////////////////////////////////

  const auto epsilon = 1.0e-8;
  const auto forder = static_cast<double>(order);
  const auto reference = 0.25 * std::pow(forder,3) * std::pow(forder-1.0,2) * (iterations+1);
  const auto checksum = prk_reduce( &(h_c[0]), &(h_c[nelems]), 0.0);
  const auto residuum = std::abs(checksum-reference)/reference;

  if (residuum < epsilon) {
#if VERBOSE
    std::cout << "Reference checksum = " << reference << "\n"
              << "Actual checksum = " << checksum << std::endl;
#endif
    std::cout << "Solution validates" << std::endl;
    auto avgtime = dgemm_time/iterations;
    auto nflops = 2.0 * std::pow(forder,3);
    std::cout << "Rate (MF/s): " << 1.0e-6 * nflops/avgtime
              << " Avg time (s): " << avgtime << std::endl;
  } else {
    std::cout << "Reference checksum = " << reference << "\n"
              << "Actual checksum = " << checksum << std::endl;
    return 1;
  }

  prk::CUDA::check( hipHostFree(h_c) );

  return 0;
}


