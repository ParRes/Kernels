#include "hip/hip_runtime.h"
///
/// Copyright (c) 2018, Intel Corporation
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions
/// are met:
///
/// * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
/// * Redistributions in binary form must reproduce the above
///       copyright notice, this list of conditions and the following
///       disclaimer in the documentation and/or other materials provided
///       with the distribution.
/// * Neither the name of Intel Corporation nor the names of its
///       contributors may be used to endorse or promote products
///       derived from this software without specific prior written
///       permission.
///
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
/// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
/// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
/// COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
/// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
/// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
/// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
/// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
/// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
/// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.

//////////////////////////////////////////////////////////////////////
///
/// NAME:    dgemm
///
/// PURPOSE: This program tests the efficiency with which a dense matrix
///          dense multiplication is carried out
///
/// USAGE:   The program takes as input the matrix order,
///          the number of times the matrix-matrix multiplication
///          is carried out, and, optionally, a tile size for matrix
///          blocking
///
///          <progname> <# iterations> <matrix order> [<batches>]
///
///          The output consists of diagnostics to make sure the
///          algorithm worked, and of timing statistics.
///
/// FUNCTIONS CALLED:
///
///          Other than OpenMP or standard C functions, the following
///          functions are used in this program:
///
///          cblasDgemm()
///          hipblasDgemmStridedBatched()
///
/// HISTORY: Written by Rob Van der Wijngaart, February 2009.
///          Converted to C++11 by Jeff Hammond, December, 2017.
///
//////////////////////////////////////////////////////////////////////

#include "prk_util.h"
#include "prk_cuda.h"

__global__ void init(int order, const int matrices, double * A, double * B, double * C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    for (int b=0; b<matrices; ++b) {
      if ((i<order) && (j<order)) {
        A[b*order*order+i*order+j] = i;
        B[b*order*order+i*order+j] = i;
        C[b*order*order+i*order+j] = 0;
      }
    }
}

__global__ void init(int order, const int matrices, double * C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    for (int b=0; b<matrices; ++b) {
      if ((i<order) && (j<order)) {
        C[b*order*order+i*order+j] = 0;
      }
    }
}

void prk_dgemm(const hipblasHandle_t & h,
               const int order,
               const int batches,
               double * A,
               double * B,
               double * C)
{
    const double alpha = 1.0;
    const double beta  = 1.0;

    for (int b=0; b<batches; ++b) {
        double * pA = &(A[b*order*order]);
        double * pB = &(B[b*order*order]);
        double * pC = &(C[b*order*order]);
        prk::CUDA::check( hipblasDgemm(h,
                                      HIPBLAS_OP_N, HIPBLAS_OP_N, // opA, opB
                                      order, order, order,      // m, n, k
                                      &alpha,                   // alpha
                                      pA, order,                // A, lda
                                      pB, order,                // B, ldb
                                      &beta,                    // beta
                                      pC, order) );             // C, ldc
    }
    prk::CUDA::check( hipDeviceSynchronize() );
}

void prk_bgemm(const hipblasHandle_t & h,
               const int order,
               const int batches,
               double * A,
               double * B,
               double * C)
{
    const double alpha = 1.0;
    const double beta  = 1.0;

    prk::CUDA::check( hipblasDgemmStridedBatched(h,
                                                HIPBLAS_OP_N, HIPBLAS_OP_N,
                                                order, order, order,
                                                &alpha,
                                                (const double *)A, order, order*order,
                                                (const double *)B, order, order*order,
                                                &beta,
                                                C, order, order*order,
                                                batches) );
    prk::CUDA::check( hipDeviceSynchronize() );

    //  hipblasStatus_t hipblasDgemmBatched(hipblasHandle_t handle,
    //                                    hipblasOperation_t transa,
    //                                    hipblasOperation_t transb,
    //                                    int m, int n, int k,
    //                                    const double          *alpha,
    //                                    const double          *Aarray[], int lda,
    //                                    const double          *Barray[], int ldb,
    //                                    const double          *beta,
    //                                    double          *Carray[], int ldc,
    //                                    int batchCount)
}

int main(int argc, char * argv[])
{
  std::cout << "Parallel Research Kernels version " << PRKVERSION << std::endl;
  std::cout << "C++11/CUBLAS Dense matrix-matrix multiplication: C += A x B" << std::endl;

  prk::CUDA::info info;
  info.print();

  //////////////////////////////////////////////////////////////////////
  /// Read and test input parameters
  //////////////////////////////////////////////////////////////////////

  int iterations;
  int order;
  int batches = 0;
  int input_copy = 0;
  try {
      if (argc < 2) {
        throw "Usage: <# iterations> <matrix order> [<batches>] [<copy input every iteration [0/1]>]";
      }

      iterations  = std::atoi(argv[1]);
      if (iterations < 1) {
        throw "ERROR: iterations must be >= 1";
      }

      order = std::atoi(argv[2]);
      if (order <= 0) {
        throw "ERROR: Matrix Order must be greater than 0";
      } else if (order > std::floor(std::sqrt(INT_MAX))) {
        throw "ERROR: matrix dimension too large - overflow risk";
      }

      if (argc>3) {
        batches = std::atoi(argv[3]);
      }

      if (argc > 4) {
        input_copy = std::atoi(argv[3]);
      }
  }
  catch (const char * e) {
    std::cout << e << std::endl;
    return 1;
  }

  std::cout << "Number of iterations = " << iterations << std::endl;
  std::cout << "Matrix order         = " << order << std::endl;
  if (batches == 0) {
      std::cout << "No batching" << std::endl;
  } else if (batches < 0) {
      std::cout << "Batch size           = " << -batches << " (loop over legacy BLAS)" << std::endl;
  } else if (batches > 0) {
      std::cout << "Batch size           = " <<  batches << " (batched BLAS)" << std::endl;
  }
  std::cout << "Input copy           = " << (input_copy ? "yes" : "no") << std::endl;

  hipblasHandle_t h;
  prk::CUDA::check( hipblasCreate(&h) );

  const int tile_size = 32;
  dim3 dimGrid(prk::divceil(order,tile_size),prk::divceil(order,tile_size),1);
  dim3 dimBlock(tile_size, tile_size, 1);

  info.checkDims(dimBlock, dimGrid);

  //////////////////////////////////////////////////////////////////////
  // Allocate space for matrices
  //////////////////////////////////////////////////////////////////////

  double dgemm_time(0);

  const int matrices = (batches==0 ? 1 : abs(batches));
  const size_t nelems = (size_t)order * (size_t)order;
  const size_t bytes = nelems * sizeof(double);

  // host buffers
  double * h_a;
  double * h_b;
  double * h_c;
  prk::CUDA::check( hipHostMalloc((void**)&h_a, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_b, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_c, matrices*bytes) );

  // device buffers
  double * d_a;
  double * d_b;
  double * d_c;
  prk::CUDA::check( hipMalloc((void**)&d_a, matrices*bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_b, matrices*bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_c, matrices*bytes) );

  if (input_copy) {

    for (int i=0; i<order; ++i) {
      for (int j=0; j<order; ++j) {
         h_a[i*order+j] = i;
         h_b[i*order+j] = i;
      }
    }

    for (int b=0; b<matrices; ++b) {
      prk::CUDA::check( hipMemcpyAsync(&(d_a[b*order*order]), h_a, bytes, hipMemcpyHostToDevice) );
      prk::CUDA::check( hipMemcpyAsync(&(d_b[b*order*order]), h_b, bytes, hipMemcpyHostToDevice) );
    }
    prk::CUDA::check( hipDeviceSynchronize() );

    init<<<dimGrid, dimBlock>>>(order, matrices, d_c);

  } else {

    init<<<dimGrid, dimBlock>>>(order, matrices, d_a, d_b, d_c);

  }

  {
    for (auto iter = 0; iter<=iterations; iter++) {

      if (iter==1) dgemm_time = prk::wtime();

      if (input_copy) {
        for (int b=0; b<matrices; ++b) {
          prk::CUDA::check( hipMemcpyAsync(&(d_a[b*order*order]), h_a, bytes, hipMemcpyHostToDevice) );
          prk::CUDA::check( hipMemcpyAsync(&(d_b[b*order*order]), h_b, bytes, hipMemcpyHostToDevice) );
        }
        prk::CUDA::check( hipDeviceSynchronize() );
      }

      if (batches == 0) {
        prk_dgemm(h, order, matrices, d_a, d_b, d_c);
      } else if (batches < 0) {
        prk_dgemm(h, order, matrices, d_a, d_b, d_c);
      } else if (batches > 0) {
        prk_bgemm(h, order, matrices, d_a, d_b, d_c);
      }
    }
    dgemm_time = prk::wtime() - dgemm_time;
  }

  // copy output back to host
  prk::CUDA::check( hipMemcpyAsync(&(h_c[0]), d_c, matrices*bytes, hipMemcpyDeviceToHost) );

  prk::CUDA::check( hipFree(d_c) );
  prk::CUDA::check( hipFree(d_b) );
  prk::CUDA::check( hipFree(d_a) );

  prk::CUDA::check( hipHostFree(h_a) );
  prk::CUDA::check( hipHostFree(h_b) );

  prk::CUDA::check( hipblasDestroy(h) );

  prk::CUDA::check( hipDeviceSynchronize() );

  //////////////////////////////////////////////////////////////////////
  /// Analyze and output results
  //////////////////////////////////////////////////////////////////////

  const double epsilon = 1.0e-8;
  const double forder = static_cast<double>(order);
  const double reference = 0.25 * std::pow(forder,3) * std::pow(forder-1.0,2) * (iterations+1);
  double residuum(0);
  for (int b=0; b<matrices; ++b) {
      const auto checksum = prk_reduce( &(h_c[b*order*order+0]), &(h_c[b*order*order+nelems]), 0.0);
      residuum += std::abs(checksum-reference)/reference;
  }
  residuum/=matrices;

  if (residuum < epsilon) {
#if VERBOSE
    std::cout << "Reference checksum = " << reference << "\n"
              << "Actual checksum = " << checksum << std::endl;
#endif
    std::cout << "Solution validates" << std::endl;
    auto avgtime = dgemm_time/iterations/matrices;
    auto nflops = 2.0 * std::pow(forder,3);
    std::cout << "Rate (MF/s): " << 1.0e-6 * nflops/avgtime
              << " Avg time (s): " << avgtime << std::endl;
  } else {
    std::cout << "Reference checksum = " << reference << "\n"
              << "Residuum           = " << residuum << std::endl;
    return 1;
  }

  prk::CUDA::check( hipHostFree(h_c) );

  return 0;
}


