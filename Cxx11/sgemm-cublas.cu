#include "hip/hip_runtime.h"
///
/// Copyright (c) 2018, Intel Corporation
/// Copyright (c) 2021, NVIDIA
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions
/// are met:
///
/// * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
/// * Redistributions in binary form must reproduce the above
///       copyright notice, this list of conditions and the following
///       disclaimer in the documentation and/or other materials provided
///       with the distribution.
/// * Neither the name of Intel Corporation nor the names of its
///       contributors may be used to endorse or promote products
///       derived from this software without specific prior written
///       permission.
///
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
/// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
/// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
/// COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
/// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
/// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
/// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
/// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
/// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
/// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.

//////////////////////////////////////////////////////////////////////
///
/// NAME:    sgemm
///
/// PURPOSE: This program tests the efficiency with which a dense matrix
///          dense multiplication is carried out
///
/// USAGE:   The program takes as input the matrix order,
///          the number of times the matrix-matrix multiplication
///          is carried out, and, optionally, a tile size for matrix
///          blocking
///
///          <progname> <# iterations> <matrix order> [<batches>]
///
///          The output consists of diagnostics to make sure the
///          algorithm worked, and of timing statistics.
///
/// FUNCTIONS CALLED:
///
///          Other than OpenMP or standard C functions, the following
///          functions are used in this program:
///
///          cblasSgemm()
///          hipblasSgemmStridedBatched()
///
/// HISTORY: Written by Rob Van der Wijngaart, February 2009.
///          Converted to C++11 by Jeff Hammond, December, 2017.
///
//////////////////////////////////////////////////////////////////////

#include "prk_util.h"
#include "prk_cuda.h"

#if 0
__global__ void init(unsigned order, float * A, float * B, float * C)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i<order) && (j<order)) {
      A[i*order+j] = i;
      B[i*order+j] = i;
      C[i*order+j] = 0;
    }
}

__global__ void init(unsigned order, float * C)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i<order) && (j<order)) {
      C[i*order+j] = 0;
    }
}
#endif

__global__ void init(int order, const int matrices, float * A, float * B, float * C)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    for (int b=0; b<matrices; ++b) {
      if ((i<order) && (j<order)) {
        A[b*order*order+i*order+j] = i;
        B[b*order*order+i*order+j] = i;
        C[b*order*order+i*order+j] = 0;
      }
    }
}

__global__ void init(int order, const int matrices, float * C)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    for (int b=0; b<matrices; ++b) {
      if ((i<order) && (j<order)) {
        C[b*order*order+i*order+j] = 0;
      }
    }
}

void prk_sgemm(const hipblasHandle_t & h,
               const int order,
               const int batches,
               float * A,
               float * B,
               float * C)
{
    const float alpha = 1.0;
    const float beta  = 1.0;

    for (int b=0; b<batches; ++b) {
        float * pA = &(A[b*order*order]);
        float * pB = &(B[b*order*order]);
        float * pC = &(C[b*order*order]);
        prk::CUDA::check( hipblasSgemm(h,
                                      HIPBLAS_OP_N, HIPBLAS_OP_N, // opA, opB
                                      order, order, order,      // m, n, k
                                      &alpha,                   // alpha
                                      pA, order,                // A, lda
                                      pB, order,                // B, ldb
                                      &beta,                    // beta
                                      pC, order) );             // C, ldc
    }
}

void prk_bgemm(const hipblasHandle_t & h,
               const int order,
               const int batches,
               float * A,
               float * B,
               float * C)
{
    const float alpha = 1.0;
    const float beta  = 1.0;

    prk::CUDA::check( hipblasSgemmStridedBatched(h,
                                                HIPBLAS_OP_N, HIPBLAS_OP_N,
                                                order, order, order,
                                                &alpha,
                                                (const float *)A, order, order*order,
                                                (const float *)B, order, order*order,
                                                &beta,
                                                C, order, order*order,
                                                batches) );

    //  hipblasStatus_t hipblasSgemmBatched(hipblasHandle_t handle,
    //                                    hipblasOperation_t transa,
    //                                    hipblasOperation_t transb,
    //                                    int m, int n, int k,
    //                                    const float          *alpha,
    //                                    const float          *Aarray[], int lda,
    //                                    const float          *Barray[], int ldb,
    //                                    const float          *beta,
    //                                    float          *Carray[], int ldc,
    //                                    int batchCount)
}

int main(int argc, char * argv[])
{
  std::cout << "Parallel Research Kernels version " << PRKVERSION << std::endl;
  std::cout << "C++11/CUBLAS Dense matrix-matrix multiplication: C += A x B" << std::endl;

  prk::CUDA::info info;
  //info.print();

  //////////////////////////////////////////////////////////////////////
  /// Read and test input parameters
  //////////////////////////////////////////////////////////////////////

  int iterations;
  int order;
  int batches = 0;
  int input_copy = 0;
  try {
      if (argc < 2) {
        throw "Usage: <# iterations> <matrix order> [<batches>] [<copy input every iteration [0/1]>]";
      }

      iterations  = std::atoi(argv[1]);
      if (iterations < 1) {
        throw "ERROR: iterations must be >= 1";
      }

      order = std::atoi(argv[2]);
      if (order <= 0) {
        throw "ERROR: Matrix Order must be greater than 0";
      } else if (order > prk::get_max_matrix_size()) {
        throw "ERROR: matrix dimension too large - overflow risk";
      }

      if (argc > 3) {
        batches = std::atoi(argv[3]);
      }

      if (argc > 4) {
        input_copy = std::atoi(argv[4]);
        if (input_copy != 0 && input_copy != 1) {
          throw "ERROR: input_copy was not 0 or 1";
        }
      }
  }
  catch (const char * e) {
    std::cout << e << std::endl;
    return 1;
  }

  std::cout << "Number of iterations = " << iterations << std::endl;
  std::cout << "Matrix order         = " << order << std::endl;
  if (batches == 0) {
      std::cout << "No batching" << std::endl;
  } else if (batches < 0) {
      std::cout << "Batch size           = " << -batches << " (loop over legacy BLAS)" << std::endl;
  } else if (batches > 0) {
      std::cout << "Batch size           = " <<  batches << " (batched BLAS)" << std::endl;
  }
  std::cout << "Input copy           = " << (input_copy ? "yes" : "no") << std::endl;

  hipblasHandle_t h;
  prk::CUDA::check( hipblasCreate(&h) );

  const int tile_size = 32;
  dim3 dimGrid(prk::divceil(order,tile_size),prk::divceil(order,tile_size),1);
  dim3 dimBlock(tile_size, tile_size, 1);

  info.checkDims(dimBlock, dimGrid);

  //////////////////////////////////////////////////////////////////////
  // Allocate space for matrices
  //////////////////////////////////////////////////////////////////////

  double gemm_time(0);

  const int matrices = (batches==0 ? 1 : abs(batches));
  const size_t nelems = (size_t)order * (size_t)order;

  // host buffers
  auto h_a = prk::CUDA::malloc_host<float>(nelems);
  auto h_b = prk::CUDA::malloc_host<float>(nelems);
  auto h_c = prk::CUDA::malloc_host<float>(matrices*nelems);

  // device buffers
  auto d_a = prk::CUDA::malloc_device<float>(matrices*nelems);
  auto d_b = prk::CUDA::malloc_device<float>(matrices*nelems);
  auto d_c = prk::CUDA::malloc_device<float>(matrices*nelems);

  if (input_copy) {
    for (int i=0; i<order; ++i) {
      for (int j=0; j<order; ++j) {
         h_a[i*order+j] = i;
         h_b[i*order+j] = i;
      }
    }

    for (int b=0; b<matrices; ++b) {
      prk::CUDA::copyH2Dasync(&(d_a[b*nelems]), h_a, nelems);
      prk::CUDA::copyH2Dasync(&(d_b[b*nelems]), h_b, nelems);
    }
    prk::CUDA::sync();

    init<<<dimGrid, dimBlock>>>(order, matrices, d_c);

  } else {

    init<<<dimGrid, dimBlock>>>(order, matrices, d_a, d_b, d_c);

  }
  prk::CUDA::sync();

  double xfer(0);
  double comp(0);
  {
    for (int iter = 0; iter<=iterations; iter++) {

      if (iter==1) {
          prk::CUDA::sync();
          gemm_time = prk::wtime();
      }

      if (input_copy) {
        double t0 = prk::wtime();
        for (int b=0; b<matrices; ++b) {
          prk::CUDA::copyH2Dasync(&(d_a[b*nelems]), h_a, nelems);
          prk::CUDA::copyH2Dasync(&(d_b[b*nelems]), h_b, nelems);
        }
        prk::CUDA::sync();
        double t1 = prk::wtime();
        if (iter==1) xfer += (t1-t0);
      }

      {
        double t0 = prk::wtime();
        if (batches > 0) {
          prk_bgemm(h, order, matrices, d_a, d_b, d_c);
        } else {
          prk_sgemm(h, order, matrices, d_a, d_b, d_c);
        }
        double t1 = prk::wtime();
        if (iter==1) comp += (t1-t0);
      }
    }
    prk::CUDA::sync();
    gemm_time = prk::wtime() - gemm_time;
  }
  std::cout << "xfer, comp = " << xfer << "," << comp << std::endl;

  // copy output back to host
  prk::CUDA::copyD2H(h_c, d_c, matrices*nelems);

  prk::CUDA::free(d_a);
  prk::CUDA::free(d_b);
  prk::CUDA::free(d_c);

  prk::CUDA::free_host(h_a);
  prk::CUDA::free_host(h_b);

  prk::CUDA::check( hipblasDestroy(h) );

  prk::CUDA::sync();

  //////////////////////////////////////////////////////////////////////
  /// Analyze and output results
  //////////////////////////////////////////////////////////////////////

  const auto epsilon = 1.0e-8;
  const auto forder = static_cast<double>(order);
  const auto reference = 0.25 * prk::pow(forder,3) * prk::pow(forder-1.0,2) * (iterations+1);
  double residuum(0);
  for (int b=0; b<matrices; ++b) {
      const auto checksum = prk::reduce( &(h_c[b*order*order+0]), &(h_c[b*order*order+nelems]), 0.0);
      residuum += std::abs(checksum-reference)/reference;
  }
  residuum /= matrices;

  if (residuum < epsilon) {
#if VERBOSE
    std::cout << "Reference checksum = " << reference << "\n"
              << "Actual checksum = " << checksum << std::endl;
#endif
    std::cout << "Solution validates" << std::endl;
    auto avgtime = gemm_time/iterations/matrices;
    auto nflops = 2.0 * prk::pow(forder,3);
    std::cout << "Rate (MF/s): " << 1.0e-6 * nflops/avgtime
              << " Avg time (s): " << avgtime << std::endl;
  } else {
    std::cout << "Reference checksum = " << reference << "\n"
              << "Residuum           = " << residuum << std::endl;
    return 1;
  }

  prk::CUDA::free_host(h_c);

  return 0;
}


