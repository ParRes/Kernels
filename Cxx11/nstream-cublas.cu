#include "hip/hip_runtime.h"
///
/// Copyright (c) 2020, Intel Corporation
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions
/// are met:
///
/// * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
/// * Redistributions in binary form must reproduce the above
///       copyright notice, this list of conditions and the following
///       disclaimer in the documentation and/or other materials provided
///       with the distribution.
/// * Neither the name of Intel Corporation nor the names of its
///       contributors may be used to endorse or promote products
///       derived from this software without specific prior written
///       permission.
///
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
/// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
/// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
/// COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
/// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
/// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
/// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
/// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
/// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
/// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.

//////////////////////////////////////////////////////////////////////
///
/// NAME:    nstream
///
/// PURPOSE: To compute memory bandwidth when adding a vector of a given
///          number of double precision values to the scalar multiple of
///          another vector of the same length, and storing the result in
///          a third vector.
///
/// USAGE:   The program takes as input the number
///          of iterations to loop over the triad vectors and
///          the length of the vectors.
///
///          <progname> <# iterations> <vector length>
///
///          The output consists of diagnostics to make sure the
///          algorithm worked, and of timing statistics.
///
/// NOTES:   Bandwidth is determined as the number of words read, plus the
///          number of words written, times the size of the words, divided
///          by the execution time. For a vector length of N, the total
///          number of words read and written is 4*N*sizeof(double).
///
/// HISTORY: This code is loosely based on the Stream benchmark by John
///          McCalpin, but does not follow all the Stream rules. Hence,
///          reported results should not be associated with Stream in
///          external publications
///
///          Converted to C++11 by Jeff Hammond, November 2017.
///
//////////////////////////////////////////////////////////////////////

#include "prk_util.h"
#include "prk_cuda.h"

int main(int argc, char * argv[])
{
  std::cout << "Parallel Research Kernels version " << PRKVERSION << std::endl;
  std::cout << "C++11/CUBLAS STREAM triad: A = B + scalar * C" << std::endl;

  prk::CUDA::info info;
  //info.print();

  //////////////////////////////////////////////////////////////////////
  /// Read and test input parameters
  //////////////////////////////////////////////////////////////////////

  int iterations;
  size_t length;
  try {
      if (argc < 3) {
        throw "Usage: <# iterations> <vector length>";
      }

      iterations  = std::atoi(argv[1]);
      if (iterations < 1) {
        throw "ERROR: iterations must be >= 1";
      }

      length = std::atol(argv[2]);
      if (length <= 0) {
        throw "ERROR: vector length must be positive";
      }
  }
  catch (const char * e) {
    std::cout << e << std::endl;
    return 1;
  }

  std::cout << "Number of iterations = " << iterations << std::endl;
  std::cout << "Vector length        = " << length << std::endl;

  hipblasHandle_t h;
  //prk::CUDA::check( cublasInit() );
  prk::CUDA::check( hipblasCreate(&h) );

  //////////////////////////////////////////////////////////////////////
  // Allocate space and perform the computation
  //////////////////////////////////////////////////////////////////////

  double nstream_time(0);

  const size_t bytes = length * sizeof(double);

  double * h_A;
  double * h_B;
  double * h_C;
  prk::CUDA::check( hipHostMalloc((void**)&h_A, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_B, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_C, bytes) );

  for (size_t i=0; i<length; ++i) {
    h_A[i] = 0;
    h_B[i] = 2;
    h_C[i] = 2;
  }

  double * d_A;
  double * d_B;
  double * d_C;
  prk::CUDA::check( hipMalloc((void**)&d_A, bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_B, bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_C, bytes) );
  prk::CUDA::check( hipMemcpy(d_A, &(h_A[0]), bytes, hipMemcpyHostToDevice) );
  prk::CUDA::check( hipMemcpy(d_B, &(h_B[0]), bytes, hipMemcpyHostToDevice) );
  prk::CUDA::check( hipMemcpy(d_C, &(h_C[0]), bytes, hipMemcpyHostToDevice) );

  double scalar(3);
  {
    for (int iter = 0; iter<=iterations; iter++) {

      if (iter==1) nstream_time = prk::wtime();

      double one(1);
      prk::CUDA::check( hipblasDaxpy(h, length,
                                    &one,        // alpha
                                    d_B, 1,      // x, incx
                                    d_A, 1) );   // y, incy
      prk::CUDA::check( hipblasDaxpy(h, length,
                                    &scalar,     // alpha
                                    d_C, 1,      // x, incx
                                    d_A, 1) );   // y, incy

      prk::CUDA::check( hipDeviceSynchronize() );
    }
    nstream_time = prk::wtime() - nstream_time;
  }

  prk::CUDA::check( hipMemcpy(&(h_A[0]), d_A, bytes, hipMemcpyDeviceToHost) );

  prk::CUDA::check( hipFree(d_C) );
  prk::CUDA::check( hipFree(d_B) );
  prk::CUDA::check( hipFree(d_A) );

  prk::CUDA::check( hipHostFree(h_B) );
  prk::CUDA::check( hipHostFree(h_C) );

  prk::CUDA::check( hipblasDestroy(h) );
  //prk::CUDA::check( cublasShutdown() );

  //////////////////////////////////////////////////////////////////////
  /// Analyze and output results
  //////////////////////////////////////////////////////////////////////

  double ar(0);
  double br(2);
  double cr(2);
  for (int i=0; i<=iterations; i++) {
      ar += br + scalar * cr;
  }

  ar *= length;

  double asum(0);
  for (int i=0; i<length; i++) {
      asum += prk::abs(h_A[i]);
  }

  prk::CUDA::check( hipHostFree(h_A) );

  double epsilon=1.e-8;
  if (prk::abs(ar-asum)/asum > epsilon) {
      std::cout << "Failed Validation on output array\n"
                << std::setprecision(16)
                << "       Expected checksum: " << ar << "\n"
                << "       Observed checksum: " << asum << std::endl;
      std::cout << "ERROR: solution did not validate" << std::endl;
      return 1;
  } else {
      std::cout << "Solution validates" << std::endl;
      double avgtime = nstream_time/iterations;
      double nbytes = 4.0 * length * sizeof(double);
      std::cout << "Rate (MB/s): " << 1.e-6*nbytes/avgtime
                << " Avg time (s): " << avgtime << std::endl;
  }

  return 0;
}


