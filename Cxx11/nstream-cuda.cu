#include "hip/hip_runtime.h"
///
/// Copyright (c) 2020, Intel Corporation
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions
/// are met:
///
/// * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
/// * Redistributions in binary form must reproduce the above
///       copyright notice, this list of conditions and the following
///       disclaimer in the documentation and/or other materials provided
///       with the distribution.
/// * Neither the name of Intel Corporation nor the names of its
///       contributors may be used to endorse or promote products
///       derived from this software without specific prior written
///       permission.
///
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
/// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
/// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
/// COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
/// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
/// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
/// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
/// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
/// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
/// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.

//////////////////////////////////////////////////////////////////////
///
/// NAME:    nstream
///
/// PURPOSE: To compute memory bandwidth when adding a vector of a given
///          number of double precision values to the scalar multiple of
///          another vector of the same length, and storing the result in
///          a third vector.
///
/// USAGE:   The program takes as input the number
///          of iterations to loop over the triad vectors and
///          the length of the vectors.
///
///          <progname> <# iterations> <vector length>
///
///          The output consists of diagnostics to make sure the
///          algorithm worked, and of timing statistics.
///
/// NOTES:   Bandwidth is determined as the number of words read, plus the
///          number of words written, times the size of the words, divided
///          by the execution time. For a vector length of N, the total
///          number of words read and written is 4*N*sizeof(double).
///
/// HISTORY: This code is loosely based on the Stream benchmark by John
///          McCalpin, but does not follow all the Stream rules. Hence,
///          reported results should not be associated with Stream in
///          external publications
///
///          Converted to C++11 by Jeff Hammond, November 2017.
///
//////////////////////////////////////////////////////////////////////

#include "prk_util.h"
#include "prk_cuda.h"

__global__ void nstream(const unsigned n, const prk_float scalar, prk_float * A, const prk_float * B, const prk_float * C)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        A[i] += B[i] + scalar * C[i];
    }
}

__global__ void nstream2(const unsigned n, const prk_float scalar, prk_float * A, const prk_float * B, const prk_float * C)
{
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        A[i] += B[i] + scalar * C[i];
    }
}

int main(int argc, char * argv[])
{
  std::cout << "Parallel Research Kernels version " << PRKVERSION << std::endl;
  std::cout << "C++11/CUDA STREAM triad: A = B + scalar * C" << std::endl;

  prk::CUDA::info info;
  info.print();

  //////////////////////////////////////////////////////////////////////
  /// Read and test input parameters
  //////////////////////////////////////////////////////////////////////

  int iterations;
  size_t length, block_size=256;
  bool grid_stride = false;
  try {
      if (argc < 3) {
        throw "Usage: <# iterations> <vector length> [<block_size>] [<grid_stride>]";
      }

      iterations  = std::atoi(argv[1]);
      if (iterations < 1) {
        throw "ERROR: iterations must be >= 1";
      }

      length = std::atol(argv[2]);
      if (length <= 0) {
        throw "ERROR: vector length must be positive";
      }

      if (argc>3) {
         block_size = std::atoi(argv[3]);
      }

      if (argc>4) {
        grid_stride = prk::parse_boolean(std::string(argv[4]));
      }
  }
  catch (const char * e) {
    std::cout << e << std::endl;
    return 1;
  }

  std::cout << "Number of iterations = " << iterations << std::endl;
  std::cout << "Vector length        = " << length << std::endl;
  std::cout << "Block size           = " << block_size << std::endl;
  std::cout << "Grid stride          = " << (grid_stride   ? "yes" : "no") << std::endl;

  dim3 dimBlock(block_size, 1, 1);
  dim3 dimGrid(prk::divceil(length,block_size), 1, 1);

  info.checkDims(dimBlock, dimGrid);

  //////////////////////////////////////////////////////////////////////
  // Allocate space and perform the computation
  //////////////////////////////////////////////////////////////////////

  double nstream_time(0);

  const size_t bytes = length * sizeof(prk_float);
  prk_float * h_A;
  prk_float * h_B;
  prk_float * h_C;
#ifndef __CORIANDERCC__
  prk::CUDA::check( hipHostMalloc((void**)&h_A, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_B, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_C, bytes) );
#else
  h_A = new prk_float[length];
  h_B = new prk_float[length];
  h_C = new prk_float[length];
#endif
  for (int i=0; i<length; ++i) {
    h_A[i] = static_cast<prk_float>(0);
    h_B[i] = static_cast<prk_float>(2);
    h_C[i] = static_cast<prk_float>(2);
  }

  prk_float * d_A;
  prk_float * d_B;
  prk_float * d_C;
  prk::CUDA::check( hipMalloc((void**)&d_A, bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_B, bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_C, bytes) );
  prk::CUDA::check( hipMemcpy(d_A, &(h_A[0]), bytes, hipMemcpyHostToDevice) );
  prk::CUDA::check( hipMemcpy(d_B, &(h_B[0]), bytes, hipMemcpyHostToDevice) );
  prk::CUDA::check( hipMemcpy(d_C, &(h_C[0]), bytes, hipMemcpyHostToDevice) );

  prk_float scalar(3);
  {
    for (int iter = 0; iter<=iterations; iter++) {

      if (iter==1) nstream_time = prk::wtime();

      if (grid_stride) {
          nstream2<<<dimGrid, dimBlock>>>(static_cast<unsigned>(length), scalar, d_A, d_B, d_C);
      } else {
          nstream<<<dimGrid, dimBlock>>>(static_cast<unsigned>(length), scalar, d_A, d_B, d_C);
      }
#ifndef __CORIANDERCC__
      // silence "ignoring hipDeviceSynchronize for now" warning
      prk::CUDA::check( hipDeviceSynchronize() );
#endif
    }
    nstream_time = prk::wtime() - nstream_time;
  }

  prk::CUDA::check( hipMemcpy(&(h_A[0]), d_A, bytes, hipMemcpyDeviceToHost) );

  prk::CUDA::check( hipFree(d_C) );
  prk::CUDA::check( hipFree(d_B) );
  prk::CUDA::check( hipFree(d_A) );

#ifndef __CORIANDERCC__
  prk::CUDA::check( hipHostFree(h_B) );
  prk::CUDA::check( hipHostFree(h_C) );
#endif

  //////////////////////////////////////////////////////////////////////
  /// Analyze and output results
  //////////////////////////////////////////////////////////////////////

  double ar(0);
  double br(2);
  double cr(2);
  for (int i=0; i<=iterations; i++) {
      ar += br + scalar * cr;
  }

  ar *= length;

  double asum(0);
  for (int i=0; i<length; i++) {
      asum += prk::abs(h_A[i]);
  }

#ifndef __CORIANDERCC__
  prk::CUDA::check( hipHostFree(h_A) );
#endif

  double epsilon=1.e-8;
  if (prk::abs(ar-asum)/asum > epsilon) {
      std::cout << "Failed Validation on output array\n"
                << std::setprecision(16)
                << "       Expected checksum: " << ar << "\n"
                << "       Observed checksum: " << asum << std::endl;
      std::cout << "ERROR: solution did not validate" << std::endl;
      return 1;
  } else {
      std::cout << "Solution validates" << std::endl;
      double avgtime = nstream_time/iterations;
      double nbytes = 4.0 * length * sizeof(prk_float);
      std::cout << "Rate (MB/s): " << 1.e-6*nbytes/avgtime
                << " Avg time (s): " << avgtime << std::endl;
  }

  return 0;
}


